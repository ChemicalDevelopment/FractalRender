extern "C" {
  
#include <hip/hip_runtime.h>
#include <stdio.h>
  #include <stdlib.h>

  __global__
  void mand_cuda_internal(unsigned char * color_output, int width, int height, int max_iter, double center_x, double center_y, double zoom) {
    int px = blockIdx.x, py = blockIdx.y;
    double x = center_x + (2 * px - width) / (zoom * width), y = center_y + (height - 2 * py) / (zoom * width);

    double sx = x, sy = y, xs = x * x, ys = y * y, tmp;

    double er = 16.0;
    double er2 = er * er;

    int col_dest = 3 * (py * width + px);

    int ci;
    for (ci = 0; ci < max_iter && xs + ys <= er2; ++ci) {
        tmp = 2 * x * y;
        x = xs - ys + sx;
        y = tmp + sy;
        xs = x * x;
        ys = y * y;
    }

    color_output[col_dest] = (3 * ci) & 0xff;
  }

  void mand_cuda(unsigned char ** color_output, int width, int height, int max_iter, double center_x, double center_y, double zoom) {
    unsigned char * _gpu_outcol;
    hipMalloc((void **)&_gpu_outcol, width * height * 3);

    dim3 dimBlock(1,1,1);
    dim3 dimGrid(width,height,1);

    mand_cuda_internal<<<dimGrid,dimBlock>>>(_gpu_outcol, width, height, max_iter, center_x, center_y, zoom);

    hipMemcpy(*color_output, _gpu_outcol, width * height * 3, hipMemcpyDeviceToHost);


    hipFree(_gpu_outcol);


  }
}
