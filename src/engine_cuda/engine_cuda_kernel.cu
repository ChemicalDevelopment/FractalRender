#include "hip/hip_runtime.h"
#include "engine_cuda/engine_cuda_helper.h"


extern "C" {
  #include <stdio.h>
  #include <stdlib.h>

  #include "engine_cuda/engine_cuda_impl.h"
  #include "fractalrender_defs.h"


  #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

  inline void gpuAssert(hipError_t code, const char *file, int line) {
     if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
     }
  }



  __global__
  void mand_cuda_internal(__cuda_gpu_fractal_t params, unsigned char * color_input, unsigned char * color_output) {
    //int px = blockIdx.x, py = blockIdx.y;
    int px = (blockIdx.x * blockDim.x) + threadIdx.x;
    int py = (blockIdx.y * blockDim.y) + threadIdx.y;

    //printf("%d:%d\n", px, py);

    double x = params.cX + (2 * px - params.px) / (params.Z * params.px), y = params.cY + (params.py - 2 * py) / (params.Z * params.px);

    double sx = x, sy = y, xs = x * x, ys = y * y, tmp;

    double er = 16.0;
    double er2 = er * er;

    int col_dest = 3 * (py * params.px + px);

    int ci;
    for (ci = 0; ci < params.max_iter && xs + ys <= er2; ++ci) {
        tmp = 2 * x * y;
        x = xs - ys + sx;
        y = tmp + sy;
        xs = x * x;
        ys = y * y;
    }

    //double tmp;
    if (params.simplecolor) {
        int color_off;
        if (ci >= params.max_iter) {
            color_off = 0;
        } else {
            color_off = 3*((int)floor(ci * params.colm + params.cold) % params.numcol);
        }
        color_output[col_dest + 0] = color_input[color_off + 0];
        color_output[col_dest + 1] = color_input[color_off + 1];
        color_output[col_dest + 2] = color_input[color_off + 2];

    } else {
        double zn = xs + ys;
        double hue;
        if (zn <= er2) {
            hue = 0;
        } else {
            hue = ci + 1.0 - log(fabs(zn)) / log(er2);
        }

        hue = hue * params.colm + params.cold;

        hue = fmod(fmod(hue, (double)params.numcol) + params.numcol, (double)params.numcol);

        tmp = hue - floor(hue);
        int color_off0 = 3 * ((int)floor(hue) % params.numcol);
        int color_off1;
        if (color_off0 >= 3 *(params.numcol - 1)) {
            color_off1 = 0;
        } else {
            color_off1 = color_off0 + 3;
        }

        color_output[col_dest + 0] = ((unsigned char)floor(tmp*color_input[color_off1 + 0]+(1-tmp)*color_input[color_off0 + 0]));
        color_output[col_dest + 1] = ((unsigned char)floor(tmp*color_input[color_off1 + 1]+(1-tmp)*color_input[color_off0 + 1]));
        color_output[col_dest + 2] = ((unsigned char)floor(tmp*color_input[color_off1 + 2]+(1-tmp)*color_input[color_off0 + 2]));
    }

    //if (ci != 0) printf("%d\n", ci);
    //cuda_color(col_dest, ci, er2, xs + ys, params, color_input, color_output);
  }

  void mand_cuda_init(__cuda_fractal_t * ret, fractal_img_t * reti) {
      ret->_gpu_fractal.px = reti->px;
      ret->_gpu_fractal.py = reti->py;

      ret->_gpu_fractal.cold = reti->color.disp;
      ret->_gpu_fractal.colm = reti->color.mult;
      ret->_gpu_fractal.simplecolor = reti->color.is_simple;

      ret->_gpu_fractal.max_iter = reti->max_iter;

      ret->_gpu_fractal.numcol = reti->color.numcol;

      ret->_gpu_fractal.cX = atof(reti->cX);
      ret->_gpu_fractal.cY = atof(reti->cY);
      ret->_gpu_fractal.Z = atof(reti->Z);

      ret->incol = reti->color.data;
      ret->outcol = reti->data;
      // ret->_gpu_outcol;

      int devID;
      hipDeviceProp_t props;

      // This will pick the best possible CUDA capable device
      devID = 0;

      //Get GPU information
      checkCudaErrors(hipSetDevice(devID));
      checkCudaErrors(hipGetDevice(&devID));
      checkCudaErrors(hipGetDeviceProperties(&props, devID));

      gpuErrchk(hipMalloc((void **)&ret->_gpu_incol, reti->color.numcol * 3));
      gpuErrchk(hipMemcpy(ret->_gpu_incol, reti->color.data, reti->color.numcol * 3, hipMemcpyHostToDevice));

      gpuErrchk(hipMalloc((void **)&ret->_gpu_outcol, reti->px * reti->py * 3));

      return;
  }

  void mand_cuda(__cuda_fractal_t * ret) {

    dim3 dimBlock(ret->cuda_size_x, ret->cuda_size_y);
    dim3 dimGrid(ret->_gpu_fractal.px/ret->cuda_size_x, ret->_gpu_fractal.py/ret->cuda_size_y);

    mand_cuda_internal<<<dimGrid,dimBlock>>>(ret->_gpu_fractal, ret->_gpu_incol, ret->_gpu_outcol);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(ret->outcol, ret->_gpu_outcol, ret->_gpu_fractal.px * ret->_gpu_fractal.py * 3, hipMemcpyDeviceToHost));

  }

  void mand_cuda_end(__cuda_fractal_t * ret) {
    if (ret->_gpu_outcol != NULL) hipFree(ret->_gpu_outcol);
    if (ret->_gpu_incol != NULL) hipFree(ret->_gpu_incol);
  }
}
