#include "hip/hip_runtime.h"

#include "fractalrender_engine_cuda_helper.h"

extern "C" {

#include <stdio.h>
#include <stdlib.h>

#include "fractalrender_types.h"
#include "fractalrender_engine_cuda.h"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(code);
   }
}


__global__
void mand_cuda_internal(fr_t fr, int width, int height, unsigned char * color_input, unsigned char * color_output) {
    //int px = blockIdx.x, py = blockIdx.y;
    int px = (blockIdx.x * blockDim.x) + threadIdx.x;
    int py = (blockIdx.y * blockDim.y) + threadIdx.y;

    //printf("%d:%d\n", px, py);

    double x = fr.prop.center_x - (width - 2 * px) / (fr.prop.zoom * fr.dim.width), y = fr.prop.center_y + (height - 2 * py) / (fr.prop.zoom * fr.dim.width);

    double sx = x, sy = y, xs = x * x, ys = y * y, tmp;


    int ri = fr.dim.mem_width * py + fr.dim.byte_depth * px;

    int ci;
    for (ci = 0; ci < fr.prop.max_iter && xs + ys <= fr.prop.er2; ++ci) {
        tmp = 2 * x * y;
        x = xs - ys + sx;
        y = tmp + sy;
        xs = x * x;
        ys = y * y;
    }

    double zn2 = xs + ys;
    if (fr.col.is_simple) {
        int colci;
        if (zn2 < fr.prop.er2) {
            colci = 0;
        } else {
            colci = (int)floor(ci * fr.col.scale + fr.col.offset);
            colci = fr.dim.byte_depth * (colci % fr.col.col_len);
        }
        color_output[ri + 0] = color_input[colci + 0];
        color_output[ri + 1] = color_input[colci + 1];
        color_output[ri + 2] = color_input[colci + 2];
        color_output[ri + 3] = color_input[colci + 3];
    } else {
        // fractional index
        double fri, mixfactor;
        if (zn2 < fr.prop.er2) {
            // index = 0, because it is inside the set
            fri = 0;
            mixfactor = 0;
            color_output[ri + 0] = color_input[0];
            color_output[ri + 1] = color_input[1];
            color_output[ri + 2] = color_input[2];
            color_output[ri + 3] = color_input[3];
        } else {
            fri = 1 + ci - log(log(sqrt(zn2))) / log(2.0);
            fri = fri * fr.col.scale + fr.col.offset;
            fri = fmod(fmod((float)fri, (float)fr.col.col_len) + fr.col.col_len, (float)fr.col.col_len);

            mixfactor = fri - floor(fri);
            int colci0, colci1;

            colci0 = (int)floor(fri);

            if (colci0 >= fr.col.col_len - 1) {
                colci1 = 0;
            } else {
                colci1 = colci0 + 1;
            }

            colci0 *= fr.dim.byte_depth;
            colci1 *= fr.dim.byte_depth;

            color_output[ri + 0] = (int)floor(mixfactor * color_input[colci1 + 0] + (1 - mixfactor) * color_input[colci0 + 0]);
            color_output[ri + 1] = (int)floor(mixfactor * color_input[colci1 + 1] + (1 - mixfactor) * color_input[colci0 + 1]);
            color_output[ri + 2] = (int)floor(mixfactor * color_input[colci1 + 2] + (1 - mixfactor) * color_input[colci0 + 2]);
            color_output[ri + 3] = (int)floor(mixfactor * color_input[colci1 + 3] + (1 - mixfactor) * color_input[colci0 + 3]);
        }
    }
}

void mand_cuda_init(cuda_fr_t * cuda_fr, fr_t * fr) {
    //cuda_fr->fr = fr;

    int devID;
    hipDeviceProp_t props;

    // This will pick the best possible CUDA capable device
    devID = 0;

    checkCudaErrors(hipSetDevice(devID));
    checkCudaErrors(hipGetDevice(&devID));
    checkCudaErrors(hipGetDeviceProperties(&props, devID));

    gpuErrchk(hipMalloc((void **)&cuda_fr->_gpu_in, fr->dim.byte_depth * fr->col.col_len));
    gpuErrchk(hipMemcpy(cuda_fr->_gpu_in, fr->col.in_col, fr->dim.byte_depth * fr->col.col_len, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void **)&cuda_fr->_gpu_out, fr->dim.mem_width * fr->dim.height));
}

void mand_cuda(cuda_fr_t * cuda_fr) {

    dim3 dimBlock(cuda_fr->dimx, cuda_fr->dimy);
    dim3 dimGrid(cuda_fr->fr->dim.width / cuda_fr->dimx, cuda_fr->fr->dim.height / cuda_fr->dimy);

    mand_cuda_internal<<<dimGrid,dimBlock>>>(*cuda_fr->fr, cuda_fr->fr->dim.width, cuda_fr->fr->dim.height, cuda_fr->_gpu_in, cuda_fr->_gpu_out);

    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    gpuErrchk(hipMemcpy(cuda_fr->fr->bitmap, cuda_fr->_gpu_out, cuda_fr->fr->dim.mem_width * cuda_fr->fr->dim.height, hipMemcpyDeviceToHost));

}

// end extern
}
